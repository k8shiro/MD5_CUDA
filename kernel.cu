#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include "md5.cuh"

int main(int argc, char* argv[])
{
	if (argc != 3) return 1;

	char* SALT = argv[1];
	char* PW = argv[2];
	char result[7];

	clock_t start, end;
	start = clock();

	/*-----���񉻂��ꂽ����-----*/
	char *dev_SALT, *dev_PW, *dev_result;

	hipMalloc((void**)&dev_SALT, strlen(SALT) *sizeof(char));
	hipMalloc((void**)&dev_PW, strlen(PW) *sizeof(char));
	hipMalloc((void**)&dev_result, 7 * sizeof(char));

	hipMemcpy(dev_SALT, SALT, strlen(SALT) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dev_PW, PW, strlen(PW) * sizeof(char), hipMemcpyHostToDevice);

	md5_brute_force<<<1000, 1000 >>>(dev_SALT, dev_PW, dev_result);
	
	hipMemcpy(result, dev_result, 7 * sizeof(char), hipMemcpyDeviceToHost);

	printf("%s\n", result);

	end = clock();
	printf("%.2f�b������܂���\n", (double)(end - start) / CLOCKS_PER_SEC);

	return 0;
}